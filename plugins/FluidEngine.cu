#include "hip/hip_runtime.h"
/* Desc: Fluid World plugin
 * Author: Andrei Haidu
 * Email: a.haidu@gmail.com
 * Date: 11 May. 2014
 */


#include "fluidix.h"
#include "FluidEngine.hh"
#include "math_functions.h"

#include "quaternion.h"

//#define DEBUG
#define GRAVITY 9.80665f 		// gravity (m/s2)
#define DT 0.001 //0.005f 				// time-step (s)
#define MAX_NR_SETS 100 		// maximum nr of sets, global arrays are created with this size

using namespace fluidix;

//////////////////////////////////////////////////
// Global variables
// Needed for external communication with Fluidix

// constants for outside communication with the fluid
struct ParticleSetConstants
{
	float particle_mass;		// particle mass (kg)
	float stiffness;			// pressure stiffness constant (k)
	float density;				// rest density (kg/m3)
	float viscosity;			// viscosity (Pa.s)
	float buoyancy;				// gas buoyancy  (m/s2)
	float threshold;    		// surface tension threshold (4.32)
	float tension;				// surface tension (N/m)
	float smth_neigh_part_nr;	// avg number of neighbors (for smoothing length)
	float h;					// smoothing length (m)
	float restitution; 			// object restitution coef (1 - elastic collision , 0 - inelastic)
};

// object set collision forces and positions, for applying forces on the rigid body physics engine
struct ObjCollisionValues
{
	xyz obj_pos;
	xyz acc_coll_forces;		// sum of forces between objects and particles
	xyz acc_coll_torque;		// sum of force position between objects and particles
};

struct Particle
{
    xyz r, v, v_hstep, f; 	// position, velocity, velocity half step, force (acceleration)
    xyz normal; 			// surface normal
    float density; 			// particle mass-density
    float tension; 			// surface tension force magnitude
};

struct Global
{
	xyz world_pos, world_size; 	// size and position of the world boundaries
	float wall_elasticity; 		// hard wall boundary elasticity
	xyz spawn_pos, spawn_size; 	// spawning particles position, and volume size
	xyz mesh_min, mesh_max; 	// spawning mesh position

	// particle set constants as global array
	ParticleSetConstants *psetdata;

	// object set collision forces and positions
	ObjCollisionValues *collision_values;

	// Default Smoothing Function, its Gradient, and its Laplacian
	inline __host__ __device__ float DefaultKernel(const float dr, const float h){
		return ((315.0f/(64.0f*PI*h*h*h*h*h*h*h*h*h))*((h*h-dr*dr)*(h*h-dr*dr)*(h*h-dr*dr)));
	}
	inline __host__ __device__ float GradDefaultKernel(const float dr, const float h){
		return ((-945.0f/(32.0f*PI*h*h*h*h*h*h*h*h*h))*(h*h-dr*dr)*(h*h-dr*dr)*dr);
	}
	inline __host__ __device__ float LaplacDefaultKernel(const float dr, const float h){
		return (-945.0f/(32.0f*PI*h*h*h*h*h*h*h*h*h))*(h*h-dr*dr)*(3*h*h-7*dr*dr);
	}
	// Pressure Smoothing Function Gradient
	inline __host__ __device__ float GradPressureKernel(const float dr, const float h){
		return (-45.0f/(PI*h*h*h*h*h*h))*(h-dr)*(h-dr);
	}
	// Viscosity Smoothing Function Laplacian
	inline __host__ __device__ float LaplacViscosityKernel(const float dr, const float h){
		return (45.0f/(PI*h*h*h*h*h*h))*(h-dr);
	}
} g;

Fluidix<> *fx;

Particle *particleSet;

//////////////////////////////////////////////////
// Function Macros

// spawn model at given point with given scale
FUNC_EACH(spawn_model,
        p.r = g.mesh_min + p.r * (g.mesh_max - g.mesh_min);
)

// Spawn fluid and init particle density
FUNC_EACH(sph_spawn,
	p.r = make_xyz(rnd_uniform() * g.spawn_size.x, 		// fluid volume X size
				   rnd_uniform() * g.spawn_size.y,		// fluid volume Y size
				   rnd_uniform() * g.spawn_size.z)		// fluid volume Z size
		+ make_xyz(g.spawn_pos.x - g.spawn_size.x/2,	// fluid X position
				   g.spawn_pos.y - g.spawn_size.y/2,	// fluid Y position
				   g.spawn_pos.z - g.spawn_size.z/2);	// fluid Z position
)

// Init SPH values
FUNC_EACH(fluid_sim_init,
	// SPH initialize each particle density
    p.density = g.psetdata[p_set].particle_mass * g.DefaultKernel(0, g.psetdata[p_set].h);

	// re-initialize values
	p.tension = 0;
	p.normal = make_xyz(0, 0, 0);
)

// add density contribution from neighbors
FUNC_PAIR(sph_density,

	float d = g.psetdata[p1_set].particle_mass * g.DefaultKernel(dr, g.psetdata[p1_set].h);

    addFloat(p1.density, d);
    addFloat(p2.density, d);
)

// apply forces on pairs of nearby particles
// pressure and viscosity force
// surface normal and tension
FUNC_PAIR(sph_pair,
	// particle pressure = K(stiffness const) * density diff
    float p1_p = g.psetdata[p1_set].stiffness * (p1.density - g.psetdata[p1_set].density);
    float p2_p = g.psetdata[p2_set].stiffness * (p2.density - g.psetdata[p2_set].density);

    // F pressure:
    xyz vec = u * (g.psetdata[p1_set].particle_mass * g.GradPressureKernel(dr, g.psetdata[p1_set].h)
    		* -(p1_p + p2_p) / (2 * p2.density));

    // F viscosity
    vec += (p2.v - p1.v) * (g.psetdata[p1_set].viscosity * g.LaplacViscosityKernel(dr, g.psetdata[p1_set].h)
    		* g.psetdata[p1_set].particle_mass / p1.density);

    // add forces to the particles
    addVector(p1.f, vec);
    addVector(p2.f, -vec);

    // inward surface normal is the gradient of the color field (4.28)
    vec = u * (g.GradDefaultKernel(dr, g.psetdata[p1_set].h)
    		* g.psetdata[p1_set].particle_mass / p2.density);

    addVector(p1.normal, vec);
    addVector(p2.normal, -vec);

    // F surface tension, a part from (4.31)
    float tens = g.LaplacDefaultKernel(dr, g.psetdata[p1_set].h)
    		* g.psetdata[p1_set].particle_mass / p2.density;

    addFloat(p1.tension, tens);
    addFloat(p2.tension, -tens);
)

// apply forces on each particle
// gravity, buoyancy
FUNC_EACH(sph_each,
	// apply gravity on the particle, this is applied without dependencies on adjacent particles
	// it is acting equally on all fluid particles
    p.f.z -= GRAVITY * p.density;

	// TODO gravity is forgotten
	// p.f.y -= BUOYANCY * (p.density - DENSITY) * GRAVITY;
    p.f.z -= g.psetdata[p_set].buoyancy * (p.density - g.psetdata[p_set].density);

    // F surface tension
    // OBS threshold and magnitude is squared (4.32, 5.17 from kelager06)
    // (should be ||n.i|| >= sqrt(thresh), we have ||n.i||^2 >= thresh, seems fine)
    float normal_magnit_sq = xyz_lensq(p.normal);
    if (normal_magnit_sq > g.psetdata[p_set].threshold) {
    	// F surface tension, the other part of (4.31)
        p.f -= p.normal * (p.tension * g.psetdata[p_set].tension / sqrtf(normal_magnit_sq));
    }
)

// SPH Semi-Implicit Euler integration and re-initialization of values
FUNC_EACH(sph_euler_integrate,
	// integration
	p.v += p.f * DT / p.density;
	p.r += p.v * DT;

	// reset forces to zero being as the timestep ends
	p.f = make_xyz(0, 0, 0);

)

// Leapfrog Integration Step
FUNC_EACH(sph_leapfrog_integrate,
    p.v_hstep += (p.f / p.density) * DT;
	p.v = p.v_hstep + (p.f / p.density) * (DT / 2);
    p.r += p.v_hstep * DT;

	// reset forces to zero being as the timestep ends
    p.f = make_xyz(0, 0, 0);
)

// Object surface collision function
FUNC_SURFACE(surface_collision,

	// compute the velocity on the unit vector direction (p.v * u)
	// compute force before changing the velocity, force = (mass * velocity) / timestep (2nd law)
	float4 p_coll_force = (p.v * g.psetdata[p_set].particle_mass) / DT;
//	float4 p_coll_force = (p.v * u * g.psetdata[p_set].particle_mass) / DT;

//	printf("p_coll_force: %f %f %f,  p_coll_force2: %f %f %f \n",
//			p_coll_force.x, p_coll_force.y, p_coll_force.z,
//			p_coll_force2.x, p_coll_force2.y, p_coll_force2.z);

	// particle position - current position of the object ^(cross_prod) particle coll_force
	float4 p_torque = (p.r -g.collision_values[p1_set].obj_pos) ^ p_coll_force;

	// particle position is projected back along the surface normal (u)
	// with the penetration depth (dr)
	p.r = p.r + dr * u;

	/* Different collision types */
	// Force based
	// p.f += 5000 * u * dr;

	// Standard Hybrid Impulse-Projection method: (4.57)
	// p.v = p.v - (1 + g.psetdata[p_set].restitution) * (p.v % u) * u;

	// Standard Hybrid Impulse-Projection method + (4.58) introducing:
 	// ratio of the penetration depth to the distance between
	// the last particle position and the penetrating position
	/* Semi-Implicit Euler Integration*/
	p.v = p.v - (1 + (g.psetdata[p1_set].restitution * dr)
			/ (DT * xyz_len(p.v))) * (p.v % u) * u;
	/* Leap Frog Integration*/
//	p.v_hstep = p.v_hstep - (1 + (g.psetdata[p_set].restitution * dr)
//			/ (DT * xyz_len(p.v_hstep))) * (p.v_hstep % u) * u;


	// add all particle forces to the global one
	addVector(g.collision_values[p1_set].acc_coll_forces, p_coll_force);

	// add torques to the global one
	addVector(g.collision_values[p1_set].acc_coll_torque, p_torque);

//	printf("set: %d coll_f[%d]: %f %f %f, g.f = %f %f %f  g.pos %f %f %f\n"
//			,p1_set, p_index, coll_force.x, coll_force.y, coll_force.z,
//			g.collision_values[p1_set].sum_coll_forces.x,
//			g.collision_values[p1_set].sum_coll_forces.y,
//			g.collision_values[p1_set].sum_coll_forces.z,
//			g.collision_values[p1_set].sum_coll_forces_pos.x,
//			g.collision_values[p1_set].sum_coll_forces_pos.y,
//			g.collision_values[p1_set].sum_coll_forces_pos.z);

)

// Surface collision with "friction", restitution coefficient
FUNC_SURFACE(static_surface_collision,
	p.r = p.r + dr * u;
	/* Semi-Implicit Euler Integration*/
	p.v = p.v - (1 + (g.psetdata[p_set].restitution * dr) / (DT * xyz_len(p.v))) * (p.v % u) * u;

	p.v *= 0.9;
)

// Surface collision with ray-triangle intersection
FUNC_COLLISION(ray_triangle_collision,
		p.r = p.r + dr * u;
		p.v = p.v - (1 + (g.psetdata[p_set].restitution * dr) / (DT * xyz_len(p.v))) * (p.v % u) * u;
		printf("Force vector: %f %f %f \n",p.f.x, p.f.y, p.f.z);
)

// Collision with Liquid World Boundary
FUNC_EACH(world_boundary,
	// X coord
	// min boundary
    if (p.r.x < g.world_pos.x - (g.world_size.x/2)) {
    	/* Semi-Implicit Euler Integration*/
    	p.v.x = g.wall_elasticity * (g.world_pos.x - (g.world_size.x/2) - p.r.x) / DT;
    	/* Leap Frog Integration*/
    	//p.v_hstep.x = g.wall_elasticity * (g.world_pos.x - (g.world_size.x/2) - p.r.x) / DT;
    	p.r.x = g.world_pos.x - (g.world_size.x/2);
    }
	// max boundary
    if (p.r.x > g.world_pos.x + (g.world_size.x/2)) {
    	/* Semi-Implicit Euler Integration*/
    	p.v.x = g.wall_elasticity * (g.world_pos.x + (g.world_size.x/2)- p.r.x) / DT;
    	/* Leap Frog Integration*/
    	//p.v_hstep.x = g.wall_elasticity * (g.world_pos.x + (g.world_size.x/2)- p.r.x) / DT;
    	p.r.x = g.world_pos.x + (g.world_size.x/2);
    }

    // Y coord
	// min boundary
    if (p.r.y < g.world_pos.y - (g.world_size.y/2)) {
    	/* Semi-Implicit Euler Integration*/
    	p.v.y = g.wall_elasticity * (g.world_pos.y - (g.world_size.y/2) - p.r.y) / DT;
    	/* Leap Frog Integration*/
    	//p.v_hstep.y = g.wall_elasticity * (g.world_pos.y - (g.world_size.y/2) - p.r.y) / DT;
    	p.r.y = g.world_pos.y - (g.world_size.y/2);
    }
	// max boundary
    if (p.r.y > g.world_pos.y + (g.world_size.y/2)) {
    	/* Semi-Implicit Euler Integration*/
    	p.v.y = g.wall_elasticity * (g.world_pos.y + (g.world_size.y/2)- p.r.y) / DT;
    	/* Leap Frog Integration*/
    	//p.v_hstep.y = g.wall_elasticity * (g.world_pos.y + (g.world_size.y/2)- p.r.y) / DT;
    	p.r.y = g.world_pos.y + (g.world_size.y/2);
    }

    // Z coord
	// min boundary
    if (p.r.z < g.world_pos.z - (g.world_size.z/2)) {
    	/* Semi-Implicit Euler Integration*/
    	p.v.z = g.wall_elasticity * (g.world_pos.z - (g.world_size.z/2) - p.r.z) / DT;
    	/* Leap Frog Integration*/
    	//p.v_hstep.z = g.wall_elasticity * (g.world_pos.z - (g.world_size.z/2) - p.r.z) / DT;
    	p.r.z = g.world_pos.z - (g.world_size.z/2);
    }
	// max boundary
    if (p.r.z > g.world_pos.z + (g.world_size.z/2)) {
    	/* Semi-Implicit Euler Integration*/
    	p.v.z = g.wall_elasticity * (g.world_pos.z + (g.world_size.z/2)- p.r.z) / DT;
    	/* Leap Frog Integration*/
    	//p.v_hstep.z = g.wall_elasticity * (g.world_pos.z + (g.world_size.z/2)- p.r.z) / DT;
    	p.r.z = g.world_pos.z + (g.world_size.z/2);
    }
)

//////////////////////////////////////////////////
FluidEngine::FluidEngine()
{
	// Initialize Fluidix
    fx = new Fluidix<>(&g);

    // Initialize members
    worldBoundariesCreated = false;

    // Init global arrays
    fx->createGlobalArray(&g.psetdata, MAX_NR_SETS * sizeof(ParticleSetConstants));

	// Init global array of object set collision forces and positions
    fx->createGlobalArray(&g.collision_values, MAX_NR_SETS * sizeof(ObjCollisionValues));
}

//////////////////////////////////////////////////
FluidEngine::~FluidEngine()
{
// TODO if destructor is called before last CUDA update it crashes badly
//	delete fx;
}

//////////////////////////////////////////////////
void FluidEngine::Init()
{

}

//////////////////////////////////////////////////
void FluidEngine::Update()
{
    fx->setTimer();

    for (IdToFluidSetMap::iterator fluid_iter = this->idToFluid_M.begin();
            fluid_iter != this->idToFluid_M.end(); fluid_iter++)
    {
    	// Init SPH values
    	fx->runEach(fluid_sim_init(),
    			fluid_iter->second->GetParticleSetId()); 				// particle Set of the Fluid

    	// add density contribution from neighbors
    	fx->runPair(sph_density(),
    			fluid_iter->second->GetParticleSetId(), 				// particle Set of the Fluid
    			fluid_iter->second->GetParticleSetId(), 				// particle Set of the Fluid
    			g.psetdata[fluid_iter->second->GetParticleSetId()].h); 	// smoothing length of of the Fluid

    	// apply forces on pairs of nearby particles, pressure, viscosity and surface tension force
    	fx->runPair(sph_pair(),
    			fluid_iter->second->GetParticleSetId(), 				// particle Set of the Fluid
    			fluid_iter->second->GetParticleSetId(), 				// particle Set of the Fluid
    			g.psetdata[fluid_iter->second->GetParticleSetId()].h);	// smoothing length of the Fluid

    	// apply forces on each particle, gravity and buoyancy
    	fx->runEach(sph_each(),
    			fluid_iter->second->GetParticleSetId()); 		// particle Set of the Fluid



    	// if world boundaries are created interact with them
    	if (this->worldBoundariesCreated)
    	{
    		fx->runEach(world_boundary(),
    				fluid_iter->second->GetParticleSetId()); 	// particle Set of the Fluid
    	}

        // interact with all STATIC Objects
        for (IdToObjectSetMap::iterator object_iter = this->idToStaticObject_M.begin();
                object_iter != this->idToStaticObject_M.end(); object_iter++)
        {
            // apply surface collision with static object
            fx->runSurface(static_surface_collision(),
                    object_iter->second->GetLinkId(), 			// link Set of the Object
                    fluid_iter->second->GetParticleSetId(),		// particle Set of the Object
                    -1);										// -1 = interact with all depth sizes

        }

        // interact with all MOVABLE Objects
        for (IdToObjectSetMap::iterator movable_object_iter = this->idToMovableObject_M.begin();
                movable_object_iter != this->idToMovableObject_M.end(); movable_object_iter++)
        {
//        	fx->runCollision(ray_triangle_collision(),
//                    movable_object_iter->second->GetLinkId(), 	// link Set of the Movable Object
//                    fluid_iter->second->GetParticleSetId(),		// particle Set of the Movable Object
//                    DT);										// time step for the ray start point r-v*dt

            // apply surface collision with movable object
            fx->runSurface(surface_collision(),
                    movable_object_iter->second->GetLinkId(), 	// link Set of the Movable Object
                    fluid_iter->second->GetParticleSetId(),		// particle Set of the Movable Object
                    -1);										// -1 = interact with all depth sizes
        }

    	// Semi Impliciy Euler integration
    	fx->runEach(sph_euler_integrate(),
    			fluid_iter->second->GetParticleSetId()); 		// particle Set of the Fluid

    	// Leapfrog integration
    	// fx->runEach(sph_leapfrog_integrate(),
    	// fluid_iter->second->GetParticleSetId()); 			// particle Set of the Fluid
    }


    // get global collision data (forces/forces position) from the gpu memory
    fx->getGlobalArray(&g.collision_values);

    // loop through all the objects to add the values to the objects and reset them
    for (IdToObjectSetMap::iterator movable_object_iter = this->idToMovableObject_M.begin();
            movable_object_iter != this->idToMovableObject_M.end(); movable_object_iter++)
    {
    	// set the sum of forces and positions from the global gpu memory
    	movable_object_iter->second->SetCollisionForceSum(
    			g.collision_values[movable_object_iter->first].acc_coll_forces);
    	movable_object_iter->second->SetCollisionForcePosSum(
    			g.collision_values[movable_object_iter->first].acc_coll_torque);

    	// reset the global values of the sums
    	g.collision_values[movable_object_iter->first].acc_coll_forces = make_xyz(0,0,0);
    	g.collision_values[movable_object_iter->first].acc_coll_torque = make_xyz(0,0,0);
    	g.collision_values[movable_object_iter->first].obj_pos =
    			movable_object_iter->second->GetWorldPosition();
    }

    // set the reseted collision data to the gpu memory
    fx->applyGlobalArray(&g.collision_values);



    // get the durration of the computation in order to compare it to the real time factor
    float comp_durr = fx->getTimer();

    printf("*GPU* Update dur: %.2f ms;\n", comp_durr);
}

//////////////////////////////////////////////////
void FluidEngine::CreateWorldBoundaries(float3 _pos, float3 _size, float _elasticity)
{
	// set flag in order to compute collisions with the world boundaries
	this->worldBoundariesCreated = true;

	// Set world position
	g.world_pos = make_xyz(_pos.x, _pos.y, _pos.z);

	// Set world boundary size
	g.world_size = make_xyz(_size.x, _size.y, _size.z);

	// Set hard wall boundary elasticity
	g.wall_elasticity = _elasticity;
}

//////////////////////////////////////////////////
void FluidEngine::AddFluidSet(float3 _spawn_pos,
								  float3 _volume_size,
								  int _nrNeighbors,
								  float _particleSize,
								  float _massDensity,
								  float _stiffness,
								  float _viscosity,
								  float _buoyancy,
								  float _surfaceTension)
{
	int particle_nr;
	float h;

	// local instance of the fluid class
	fluidix::FluidSet *fluidSet =  new fluidix::FluidSet();

	// set fluid volume (m3)
	fluidSet->SetVolume(_volume_size.x * _volume_size.y * _volume_size.z);

	// set particle size (m) as side of a cube
	fluidSet->SetParticleSize(_particleSize);

	// compute and set particle numbers
	particle_nr = fluidSet->GetVolume() / fluidSet->GetParticleVolume();
	fluidSet->SetParticleNr(particle_nr);

	// set avg number of neighbors (for smoothing length)
	fluidSet->SetSmoothingNeighborsNr(_nrNeighbors);

	// TODO if GetVolume is the particle volume, or the whole fluid
	// set smoothing length
	h = powf(3.0f* fluidSet->GetVolume() * fluidSet->GetSmoothingNeighboursNr() /
			(4.0f * PI * fluidSet->GetParticleNr()), 0.333f);
//	h = powf((0.75f/PI) * 20 * 0.02f / 998.29f, 0.333f);

	fluidSet->SetSmoothingLength(h);

	// set fluid density
	fluidSet->SetDensity(_massDensity);

	// set fluid stiffness
	fluidSet->SetStiffness(_stiffness);

	// set fluid viscosity
	fluidSet->SetViscosity(_viscosity);

	// set fluid buoyancu
	fluidSet->SetBuoyancy(_buoyancy);

	// set fluid surface tension
	fluidSet->SetSurfaceTension(_surfaceTension);

	// set fluid surface tension threshold
	fluidSet->SetSurfTensThreshold(_massDensity / _nrNeighbors);

	// set particle mass
	fluidSet->SetParticleMass(fluidSet->GetDensity() * fluidSet->GetVolume() /
			fluidSet->GetParticleNr());


	// get unique Id and create particle set
	int unique_id = fx->createParticleSet(particle_nr);

	// set ParticleSet unique ID
	fluidSet->SetParticleSetId(unique_id);

	// add particle set to the map
	this->idToFluid_M[unique_id] = fluidSet;


	// Add the Fluid parameters to the global array
	g.psetdata[unique_id].particle_mass = fluidSet->GetParticleMass();

	g.psetdata[unique_id].stiffness = fluidSet->GetStiffness();

	g.psetdata[unique_id].density = fluidSet->GetDensity();

	g.psetdata[unique_id].viscosity = fluidSet->GetViscosity();

	g.psetdata[unique_id].buoyancy = fluidSet->GetBuoyancy();

	g.psetdata[unique_id].tension = fluidSet->GetSurfaceTension();

	g.psetdata[unique_id].threshold = fluidSet->GetSurfTensThreshold();

	g.psetdata[unique_id].h = fluidSet->GetSmoothingLength();

	fx->applyGlobalArray(&g.psetdata);

	// set spawning position
    g.spawn_pos = make_xyz(_spawn_pos.x, _spawn_pos.y, _spawn_pos.z);

    // set spawning area
    g.spawn_size  = make_xyz(_volume_size.x, _volume_size.y, _volume_size.z);

    // Spawn fluid and init particle density for SPH
    fx->runEach(sph_spawn(), unique_id);

    // Print Info about the Fluid
    std::cout << "** Fluid set nr: " << unique_id <<", parameters:" << std::endl;
    std::cout << "Particle Nr: " << particle_nr << std::endl;
    std::cout << "Neighbours: " << _nrNeighbors << std::endl;
    std::cout << "Particle Mass: " << fluidSet->GetParticleMass() << std::endl;
    std::cout << "Stiffness: " << fluidSet->GetStiffness() << std::endl;
    std::cout << "Mass Density: " << fluidSet->GetDensity() << std::endl;
    std::cout << "Viscosity: " << fluidSet->GetViscosity() << std::endl;
    std::cout << "Buoyancy: " << fluidSet->GetBuoyancy() << std::endl;
    std::cout << "Surface Tension: " << fluidSet->GetSurfaceTension() << std::endl;
    std::cout << "Surface Tension Threshold (squared): " << fluidSet->GetSurfTensThreshold() << std::endl;
    std::cout << "Smoothing length (h): " << fluidSet->GetSmoothingLength() << std::endl;

}

//////////////////////////////////////////////////
void FluidEngine::AddFluidSet(float3 _spawn_pos,
								  float3 _volume_size,
								  int _particle_nr,
								  int _nrNeighbors,
								  float _particleSize,
								  float _massDensity,
								  float _stiffness,
								  float _viscosity,
								  float _buoyancy,
								  float _surfaceTension)
{
	float h;

	// local instance of the fluid class
	fluidix::FluidSet *fluidSet =  new fluidix::FluidSet();

	// set fluid volume (m3)
	fluidSet->SetVolume(_volume_size.x * _volume_size.y * _volume_size.z);

	// set particle size (m) as side of a cube
	fluidSet->SetParticleSize(_particleSize);

	// set particle numbers
	fluidSet->SetParticleNr(_particle_nr);

	// set avg number of neighbors (for smoothing length)
	fluidSet->SetSmoothingNeighborsNr(_nrNeighbors);

	// TODO if GetVolume is the particle volume, or the whole fluid
	// set smoothing length
	h = powf(3.0f* (fluidSet->GetParticleVolume()*_particle_nr) * fluidSet->GetSmoothingNeighboursNr() /
			(4.0f * PI * fluidSet->GetParticleNr()), 0.333f);
//	h = powf((0.75f/PI) * 20 * 0.02f / 998.29f, 0.333f);

	fluidSet->SetSmoothingLength(h);

	// set fluid density
	fluidSet->SetDensity(_massDensity);

	// set fluid stiffness
	fluidSet->SetStiffness(_stiffness);

	// set fluid viscosity
	fluidSet->SetViscosity(_viscosity);

	// set fluid buoyancy
	fluidSet->SetBuoyancy(_buoyancy);

	// set fluid surface tension
	fluidSet->SetSurfaceTension(_surfaceTension);

	// set fluid surface tension threshold
	fluidSet->SetSurfTensThreshold(_massDensity / _nrNeighbors);

	// TODO compute particle mass for custom particle nr as well
	// set particle mass
//	fluidSet->SetParticleMass(fluidSet->GetDensity() * fluidSet->GetVolume() /
//			fluidSet->GetParticleNr());
	fluidSet->SetParticleMass(fluidSet->GetDensity() * fluidSet->GetParticleVolume()*_particle_nr /
			fluidSet->GetParticleNr());

	// get unique Id and create particle set
	int unique_id = fx->createParticleSet(_particle_nr);

	// set ParticleSet unique ID
	fluidSet->SetParticleSetId(unique_id);

	// add particle set to the map
	this->idToFluid_M[unique_id] = fluidSet;


	// Add the Fluid parameters to the global array
	g.psetdata[unique_id].particle_mass = fluidSet->GetParticleMass();

	g.psetdata[unique_id].stiffness = fluidSet->GetStiffness();

	g.psetdata[unique_id].density = fluidSet->GetDensity();

	g.psetdata[unique_id].viscosity = fluidSet->GetViscosity();

	g.psetdata[unique_id].buoyancy = fluidSet->GetBuoyancy();

	g.psetdata[unique_id].tension = fluidSet->GetSurfaceTension();

	g.psetdata[unique_id].threshold = fluidSet->GetSurfTensThreshold();

	g.psetdata[unique_id].h = fluidSet->GetSmoothingLength();

	fx->applyGlobalArray(&g.psetdata);

	// set spawning position
    g.spawn_pos = make_xyz(_spawn_pos.x, _spawn_pos.y, _spawn_pos.z);

    // set spawning area
    g.spawn_size  = make_xyz(_volume_size.x, _volume_size.y, _volume_size.z);

    // Spawn fluid and init particle density for SPH
    fx->runEach(sph_spawn(), unique_id);

    // Print Info about the Fluid
    std::cout << "** Fluid set nr: " << unique_id <<", parameters:" << std::endl;
    std::cout << "Particle Nr: " << _particle_nr << std::endl;
    std::cout << "Neighbours: " << _nrNeighbors << std::endl;
    std::cout << "Particle Mass: " << fluidSet->GetParticleMass() << std::endl;
    std::cout << "Stiffness: " << fluidSet->GetStiffness() << std::endl;
    std::cout << "Mass Density: " << fluidSet->GetDensity() << std::endl;
    std::cout << "Viscosity: " << fluidSet->GetViscosity() << std::endl;
    std::cout << "Buoyancy: " << fluidSet->GetBuoyancy() << std::endl;
    std::cout << "Surface Tension: " << fluidSet->GetSurfaceTension() << std::endl;
    std::cout << "Surface Tension Threshold (squared): " << fluidSet->GetSurfTensThreshold() << std::endl;
    std::cout << "Smoothing length (h): " << fluidSet->GetSmoothingLength() << std::endl;

}

//////////////////////////////////////////////////
FluidSet FluidEngine::GetFluidSet(int _id)
{
	return *this->idToFluid_M.find(_id)->second;
}

//////////////////////////////////////////////////
void FluidEngine::AddStaticObject(std::string _path,
							float3 _spawn_pos,
							float3 _scale,
							float _restitution_coeff)
{
    // local instance of the object class
    fluidix::ObjectSet *objectSet = new fluidix::ObjectSet();

    // import model and set particle/link set unique IDs
    int2 object_ids = fx->importModel(_path.c_str());

    // set Objects Particle and Link Set IDs
    // .x = particle set index , .y link set index
    objectSet->SetParticleAndLinkSetId(object_ids.x, object_ids.y);

    // save the initial position of the object
    objectSet->SetWorldPosition(_spawn_pos);

    // TODO hardcoded
    // save the initial orientation of the object
    objectSet->SetWorldOrientation(Quaternion(1,0,0,0));

    // set restitution coefficient, // 1 - elastic collision , 0 - inelastic
    objectSet->SetRestitutionCoef(_restitution_coeff); //TODO hardcoded

    // add Object to the map, unique Id is the Particle Set ID
    this->idToStaticObject_M[object_ids.x] = objectSet;

    // set the restitution coefficient for this object
	g.psetdata[object_ids.x].restitution = objectSet->GetRestitutionCoef();
	fx->applyGlobalArray(&g.psetdata);


    // Set mesh spawning position
    g.mesh_min = make_xyz(_spawn_pos.x, _spawn_pos.y, _spawn_pos.z);
    g.mesh_max = make_xyz(_spawn_pos.x + _scale.x, _spawn_pos.y + _scale.y, _spawn_pos.z + _scale.z);

    // .x = particle set index , .y link set index
    fx->runEach(spawn_model(), object_ids.x);
}

//////////////////////////////////////////////////
int2 FluidEngine::AddMovableObject(std::string _path,
		float3 _spawn_pos,
		float4 _orientation,
		float3 _scale,
		float _restitution_coeff)
{
    // local instance of the object class
    fluidix::ObjectSet *objectSet = new fluidix::ObjectSet();

    // import model and set particle/link set unique IDs
    int2 id_set = fx->importModel(_path.c_str());

    // set Objects Particle and Link Set IDs
    // .x = particle set index , .y link set index
    objectSet->SetParticleAndLinkSetId(id_set.x, id_set.y);

    // save the initial position of the object
    objectSet->SetWorldPosition(_spawn_pos);

    // TODO check why initial orientation has to be ht unit one, otherwise it's mirrored
    // save the initial orientation of the object
    objectSet->SetWorldOrientation(Quaternion(1,0,0,0));
//    objectSet->SetWorldOrientation(Quaternion(_orientation));

    // set restitution coefficient, // 1 - elastic collision , 0 - inelastic
    objectSet->SetRestitutionCoef(_restitution_coeff);

    // add Object to the map, unique Id is the Particle Set ID
    this->idToMovableObject_M[id_set.x] = objectSet;

    // set the restitution coefficient for this object
	g.psetdata[id_set.x].restitution = objectSet->GetRestitutionCoef();
	fx->applyGlobalArray(&g.psetdata);


    // Set mesh spawning position
    g.mesh_min = make_xyz(_spawn_pos.x, _spawn_pos.y, _spawn_pos.z);
    g.mesh_max = make_xyz(_spawn_pos.x + _scale.x, _spawn_pos.y + _scale.y, _spawn_pos.z + _scale.z);

    // .x = particle set index , .y link set index
    fx->runEach(spawn_model(), id_set.x);

    return id_set;
}

//////////////////////////////////////////////////
int2 FluidEngine::AddMovableBox(float3 _spawn_pos,
		float4 _orientation,
		float3 _size,
		float _restitution_coeff)
{
    // local instance of the object class
    fluidix::ObjectSet *objectSet = new fluidix::ObjectSet();

    // unique ID of the particle/link Set
    // .x = particle set index , .y link set index
    int2 box_ids;

    // manually define the vertices of the box shape
    box_ids.x = fx->createParticleSet(8);
    // get the array of particles from the box
    Particle *p = fx->getParticleArray(box_ids.x);

    // create a unit box
	p[0].r = make_xyz(-0.5, -0.5, -0.5);
	p[1].r = make_xyz(-0.5, -0.5,  0.5);
	p[2].r = make_xyz(-0.5,  0.5, -0.5);
	p[3].r = make_xyz(-0.5,  0.5,  0.5);
	p[4].r = make_xyz( 0.5, -0.5, -0.5);
	p[5].r = make_xyz( 0.5, -0.5,  0.5);
	p[6].r = make_xyz( 0.5,  0.5, -0.5);
	p[7].r = make_xyz( 0.5,  0.5,  0.5);

	// apply changes to the array
	fx->applyParticleArray(box_ids.x);

	// define links to form triangles
	box_ids.y = fx->createLinkSet();

	fx->addLink(box_ids.y, box_ids.x, 0, box_ids.x, 1);
	fx->addLink(box_ids.y, box_ids.x, 2, box_ids.x, 3);
	fx->addLink(box_ids.y, box_ids.x, 0, box_ids.x, 2);
	fx->addLink(box_ids.y, box_ids.x, 1, box_ids.x, 3);
	fx->addLink(box_ids.y, box_ids.x, 4, box_ids.x, 5);
	fx->addLink(box_ids.y, box_ids.x, 6, box_ids.x, 7);
	fx->addLink(box_ids.y, box_ids.x, 4, box_ids.x, 6);
	fx->addLink(box_ids.y, box_ids.x, 5, box_ids.x, 7);
	fx->addLink(box_ids.y, box_ids.x, 0, box_ids.x, 4);
	fx->addLink(box_ids.y, box_ids.x, 1, box_ids.x, 5);
	fx->addLink(box_ids.y, box_ids.x, 2, box_ids.x, 6);
	fx->addLink(box_ids.y, box_ids.x, 3, box_ids.x, 7);
	fx->addLink(box_ids.y, box_ids.x, 0, box_ids.x, 3);
	fx->addLink(box_ids.y, box_ids.x, 4, box_ids.x, 7);
	fx->addLink(box_ids.y, box_ids.x, 0, box_ids.x, 5);
	fx->addLink(box_ids.y, box_ids.x, 2, box_ids.x, 7);
	fx->addLink(box_ids.y, box_ids.x, 0, box_ids.x, 6);
	fx->addLink(box_ids.y, box_ids.x, 1, box_ids.x, 7);


    // set Objects Particle and Link Set IDs
    // .x = particle set index , .y link set index
    objectSet->SetParticleAndLinkSetId(box_ids);

    // save the initial position of the object
    objectSet->SetWorldPosition(_spawn_pos);

    // TODO check why initial orientation has to be ht unit one, otherwise it's mirrored
    // save the initial orientation of the object
    objectSet->SetWorldOrientation(Quaternion(1,0,0,0));
//    objectSet->SetWorldOrientation(Quaternion(_orientation));

    // set restitution coefficient, // 1 - elastic collision , 0 - inelastic
    objectSet->SetRestitutionCoef(_restitution_coeff);

    // add Object to the map, unique Id is the Particle Set ID
    this->idToMovableObject_M[box_ids.x] = objectSet;

    // set the restitution coefficient for this object
	g.psetdata[box_ids.x].restitution = objectSet->GetRestitutionCoef();
	fx->applyGlobalArray(&g.psetdata);


    // Set box spawning position and size
    g.mesh_min = make_xyz(_spawn_pos.x, _spawn_pos.y, _spawn_pos.z);
    g.mesh_max = make_xyz(_spawn_pos.x + _size.x, _spawn_pos.y + _size.y, _spawn_pos.z + _size.z);

    // .x = particle set index , .y link set index
    fx->runEach(spawn_model(), box_ids.x);

    return box_ids;
}

//////////////////////////////////////////////////
void FluidEngine::GetParticlePositions(int _setId,
    		std::vector<float3> &_particlePositions)
{
    // get the particle set
    particleSet = fx->getParticleArray(_setId);

    // get the nr of particles
    int p_count = fx->getParticleCount(_setId);

    // write every particle postition to the vector
    for (int p_i = 0; p_i < p_count; ++p_i)
    {
    	_particlePositions[p_i].x = particleSet[p_i].r.x;
    	_particlePositions[p_i].y = particleSet[p_i].r.y;
    	_particlePositions[p_i].z = particleSet[p_i].r.z;
    }
}

//////////////////////////////////////////////////
int FluidEngine::GetFluidSetCount()
{
	return this->idToFluid_M.size();
}

//////////////////////////////////////////////////
int FluidEngine::GetObjectSetCount()
{
    return this->idToStaticObject_M.size();
}

//////////////////////////////////////////////////
int FluidEngine::GetMovableObjectSetCount()
{
    return this->idToMovableObject_M.size();
}

//////////////////////////////////////////////////
int FluidEngine::GetParticleCount(int _setId)
{
	return fx->getParticleCount(_setId);
}

//////////////////////////////////////////////////
std::vector<int> FluidEngine::GetFluidSetIDs()
{
	std::vector<int> IDs;

    for (IdToFluidSetMap::iterator iter = this->idToFluid_M.begin();
    		iter != this->idToFluid_M.end(); iter++)
    {
    	IDs.push_back(iter->second->GetParticleSetId());
    }

	return IDs;
}

//////////////////////////////////////////////////
std::vector<int> FluidEngine::GetObjectParticleSetIDs()
{
    std::vector<int> IDs;

    for (IdToObjectSetMap::iterator iter = this->idToStaticObject_M.begin();
            iter != this->idToStaticObject_M.end(); iter++)
    {
        IDs.push_back(iter->second->GetParticleSetId());
    }

    return IDs;
}

//////////////////////////////////////////////////
std::vector<int> FluidEngine::GetMovableObjectParticleSetIDs()
{
    std::vector<int> IDs;

    for (IdToObjectSetMap::iterator iter = this->idToMovableObject_M.begin();
            iter != this->idToMovableObject_M.end(); iter++)
    {
        IDs.push_back(iter->second->GetParticleSetId());
    }

    return IDs;
}

//////////////////////////////////////////////////
float3 FluidEngine::GetObjectCollisionForceSum(int _setId)
{
	xyz sum_force = this->idToMovableObject_M[_setId]->GetCollisionForceSum();

	return make_float3(sum_force.x, sum_force.y, sum_force.z);
}

//////////////////////////////////////////////////
float3 FluidEngine::GetObjectCollisionForcePosSum(int _setId)
{
	xyz sum_force_pos = this->idToMovableObject_M[_setId]->GetCollisionForcePosSum();

	return make_float3(sum_force_pos.x, sum_force_pos.y, sum_force_pos.z);
}

//////////////////////////////////////////////////
void FluidEngine::SetObjectPosition(int _setId, float4 _position)
{
    // get the particle set
    particleSet = fx->getParticleArray(_setId);

    // calculate the relative position
    float4 diff = _position - this->idToMovableObject_M[_setId]->GetWorldPosition();

    // get number of particles
    int p_count = fx->getParticleCount(_setId);

    // iterate through the particle set
    for (int i = 0; i < p_count; ++i)
    {
        particleSet[i].r += diff;
    }

    // update object new position
    this->idToMovableObject_M[_setId]->SetWorldPosition(_position);

    // apply changes made to particles outside of an interaction
    fx->applyParticleArray(_setId);

}

//////////////////////////////////////////////////
void FluidEngine::SetObjectOrientation(int _setId, Quaternion _quat)
{
	// get the world position, to translate it to the center and then back
	xyz world_pos = this->idToMovableObject_M[_setId]->GetWorldPosition();

	// the new quat * the inverse of the last position so it takes it back to the init position
	Quaternion quat = _quat * this->idToMovableObject_M[_setId]->GetWorldOrientation().GetInverse();

    // get the particle set
    particleSet = fx->getParticleArray(_setId);

    // get number of particles
    int p_count = fx->getParticleCount(_setId);

    // iterate through the particle set
    for (int i = 0; i < p_count; ++i)
    {
    	// rotate the translated position vector
    	particleSet[i].r = quat.RotateVector(particleSet[i].r - world_pos);

    	// translate back the position vector by adding back its original position
    	particleSet[i].r += world_pos;

    }

    // update object new position
    this->idToMovableObject_M[_setId]->SetWorldOrientation(_quat);

    // apply changes made to particles outside of an interaction
    fx->applyParticleArray(_setId);

}

//////////////////////////////////////////////////
void FluidEngine::SetObjectOrientation(int _setId, float _r, float _p, float _y)
{
	Quaternion quat(_r, _p, _y);

	this->SetObjectOrientation(_setId, quat);
}

// TODO overload method with different parameter types (eg, float4 instead of quat)
//////////////////////////////////////////////////
void FluidEngine::SetObjectPose(int _setId, float4 _position, Quaternion _quat)
{
	// get the world position, to translate it to the center and then back
	xyz world_pos = this->idToMovableObject_M[_setId]->GetWorldPosition();

	// the new quat * the inverse of the last position so it takes it back to the init position
	Quaternion quat = _quat * this->idToMovableObject_M[_setId]->GetWorldOrientation().GetInverse();

	// get the particle set
    particleSet = fx->getParticleArray(_setId);

    // get number of particles
    int p_count = fx->getParticleCount(_setId);

    // iterate through the particle set
    for (int i = 0; i < p_count; ++i)
    {
    	// rotate the translated position vector
    	particleSet[i].r = quat.RotateVector(particleSet[i].r - world_pos);

    	// translate back the position to the sensor value
    	particleSet[i].r += _position;
    }

    // update object new position
    this->idToMovableObject_M[_setId]->SetWorldPosition(_position);

    // update object new position
    this->idToMovableObject_M[_setId]->SetWorldOrientation(_quat);

    // apply changes made to particles outside of an interaction
    fx->applyParticleArray(_setId);
}




